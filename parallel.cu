
#include <hip/hip_runtime.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <chrono>

using namespace std;

void sieveOfEratosthenes(int *sieveInput, int max) {
    for(int i = 0; i < max-1; ++i) {
        sieveInput[i] = i+2;
    }
    for(int j= 0; j < max-1; ++j) {
        // only begin sieve round if starting point is not 0
        if(sieveInput[j]) {
            for(int k = j+sieveInput[j]; k < max-1; k += sieveInput[j]) {
                if(sieveInput[k]) {
                    sieveInput[k] = 0;
                }
            }
        }
    }
}

// remove zeroes from sieve, will only contain prime numbers
void compressSieveOfEratosthenes(int *uncompSieve, int *compSieve, int uncompressedSieveSize) {
    int counter = 0;
    for(int i = 0; i < uncompressedSieveSize; ++i) {
        if(uncompSieve[i] != 0) {
            compSieve[counter] = uncompSieve[i];
            counter++;
        }
    }
}

void generateSieve(int *uncompSieve, int maxPrime) {
    sieveOfEratosthenes(uncompSieve, maxPrime);
}

void generateCompressedSieve(int *uncompSieve, int *compSieve, int maxPrime) {
    compressSieveOfEratosthenes(uncompSieve, compSieve, maxPrime);
}

// compressed size will be used in main and kernel functions
int getCompressedSize(int *sieve, int maxPrime) {
    int compressedSize = 0;
    for(int i = 0; i < maxPrime; ++i) {
        if(sieve[i] != 0) {
            compressedSize++;
        }
    }

    return compressedSize - 1;
}

__device__ int getRandomNumber(int idx, int min, int max, hiprandState state) {
    hiprand_init(clock64(), idx, 0, &state);
    int difference = max - min + 1;
    int randomNum = (float)hiprand_uniform(&state)*difference + min;

    return randomNum;
}

// generate numerical representation of word
__device__ int generateWordNum(int size, int idx, hiprandState state) {
    hiprand_init(clock64(), idx, 0, &state);
    int wordNum = 0;
    int counter = 0;
    for(int j = 0; j < size; ++j) {
        int randomNum = getRandomNumber(idx, 1, 26, state);
        wordNum = wordNum + randomNum * pow(27, counter);
        counter++;
    }

    return wordNum;
}

__device__ int extendedEuclidGcd(long int first, long int second, long int *x, long int *y) {
    if(!first) {
        *x = 0;
        *y = 1;
        return second;
    }

    long int x1;
    long int y1;
    int gcdResult = extendedEuclidGcd(second%first, first, &x1, &y1);

    *x = y1 - (floorf(second/first)) * x1;
    *y = x1;

    return gcdResult;
}

__device__ long int euclidGcd(long int first, long int second) {
    if(!second) {
        return first;
    } else {
        long int remainder = first%second;
        return euclidGcd(second, remainder);
    }
}

// e is an odd number that is relatively prime to Phi-n
__device__ int generateE(int idx, hiprandState state, long int phiN) {
    int e = 2;
    while(e == 2 || e%2 == 0 || euclidGcd(e, phiN) != 1) {
        e = getRandomNumber(idx, 100, 500, state);
    }

    return e;
}

// calculates modulus of exponentiation without overflow
__device__ int exponentiationRemainder(long int a, long int b, long int c) {
    int result = 1;

    while(b > 0) {
        if(b%2 == 1) {
            result = (result*a)%c;
        }
        b = floorf(b/2);
        a = (a*a)%c;
    }

    return result;
}

__device__ int encrypt(long int num, long int e, long int n) {
    int encryptedNum = exponentiationRemainder(num, e, n);

    return encryptedNum;
}

__device__ int decrypt(long int num, long int e, long int phiN, long int n) {
    long int d;
    long int d1;
    long int gcd;
    gcd = extendedEuclidGcd(e, phiN, &d, &d1);
    int decryptedNum;
    if(d > 0) {
        decryptedNum = exponentiationRemainder(num, d, n);
    } else {
        // special case, use d + Phi-n as exponent
        decryptedNum = exponentiationRemainder(num, d + phiN, n);
    }

    return decryptedNum;
}

__global__ void run(int *deviceSieve, int compressedSize, int *deviceDecryption) {
    int idx = threadIdx.x + blockIdx.x*blockDim.x;

    // initialize state...used for curand
    hiprandState state;
    hiprand_init(clock64(), idx, 0, &state);

    int wordNum = generateWordNum(4, idx, state);

    // max and min index limits for sieve of eratosthenes
    int maxLimit = floorf(compressedSize/10);
    int minLimit = maxLimit - floorf(maxLimit/10);
    int randomNumberOne = getRandomNumber(idx, minLimit, maxLimit, state);
    int randomNumberTwo = getRandomNumber(idx, minLimit, maxLimit, state);
    while(randomNumberOne == randomNumberTwo) {
        randomNumberTwo = getRandomNumber(idx, minLimit, maxLimit, state);
    }
    int p = deviceSieve[randomNumberOne];
    int q = deviceSieve[randomNumberTwo];

    long int n = p*q;
    long int phiN = (p-1)*(q-1);
    long int e = generateE(idx, state, phiN);

    int encrypted = encrypt(wordNum, e, n);
    int decrypted = decrypt(encrypted, e, phiN, n);

    // keep the decryption idx value at 0 if encryption/decryption failed
    if(wordNum == decrypted) {
        deviceDecryption[idx] = wordNum;
    }
}

int main(void) {
    // get number of blocks for execution
    int numBlocks;
    cout << "Enter number of blocks: ";
    cin >> numBlocks;

    int maxPrime = 50000;
    int sieve[maxPrime];
    generateSieve(sieve, maxPrime);
    int compressedSize = getCompressedSize(sieve, maxPrime);
    int sieveOfEratosthenes[compressedSize];
    generateCompressedSieve(sieve, sieveOfEratosthenes, maxPrime);

    int *deviceSieve;
    hipMalloc(&deviceSieve, sizeof(int)*compressedSize);
    hipMemcpy(deviceSieve, sieveOfEratosthenes, sizeof(int)*compressedSize, hipMemcpyHostToDevice);

    // create decryption array and initialize values to 0
    int *decryption = (int*)malloc(1024*numBlocks*sizeof(int));
    for(int i = 0; i < 1024*numBlocks; ++i) {
        decryption[i] = 0;
    }
    int *deviceDecryption;
    hipMalloc(&deviceDecryption, sizeof(int)*1024*numBlocks);
    hipMemcpy(deviceDecryption, decryption, sizeof(int)*1024*numBlocks, hipMemcpyHostToDevice);

    auto start = chrono::high_resolution_clock::now();
    run<<<numBlocks,1024>>>(deviceSieve, compressedSize, deviceDecryption);
    hipDeviceSynchronize();
    auto end = chrono::high_resolution_clock::now();
    auto duration = chrono::duration_cast<chrono::milliseconds>(end-start);

    hipMemcpy(decryption, deviceDecryption, sizeof(int)*1024*numBlocks, hipMemcpyDeviceToHost);

    // count successes and failures
    int success = 0;
    int failure = 0;
    for(int j = 0; j < 1024*numBlocks; ++j) {
        if(decryption[j]) {
            success++;
        } else {
            failure++;
        }
    }

    // display results to user
    cout << "SUCCESS: " + to_string(success) + "\n";
    cout << "FAILURE: " + to_string(failure) + "\n";
    cout << "TOTAL TIME: " + to_string((float)duration.count()) + " MILLISECONDS" + "\n";

    // free memory at end of execution
    hipFree(deviceSieve);
    hipFree(deviceDecryption);
    free(decryption);

    return 1;
}